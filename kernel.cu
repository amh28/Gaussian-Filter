#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "CImg.h"
#include <math.h>
#include <chrono>

using namespace std;
using namespace cimg_library;


typedef CImg<unsigned char> IC;
typedef CImg<float> IF;
typedef float F;
typedef int I;


//Calculates the borders of an image using a convolution mask(Sobel operator)
IF convolution(IC img, int mask[3][3])
{
	IF imgt(img.width(), img.height());
	for (int i = 1; i<img.width() - 1; i++)
		for (int j = 1; j<img.height() - 1; j++)
		{
			float cont = 0.0;
			for (int k = -1; k <= 1; k++)
				for (int k1 = -1; k1 <= 1; k1++)
					cont += (img(i + k1, j + k)*mask[k1 + 1][k + 1]);
			float dim = 9 * 1.0;
			cont *= 1.0;
			cont /= dim;
			imgt(i, j) = cont;
		}
	//img.get_normalize(0,255);
	return imgt;
}

//Transforms an image into a vector
float* create_fils(IF img, int Width, int Height)
{
	float * fils = (float*)malloc(Height*Width*sizeof(float));
	int cont = 0;
	for (int i = 0; i < Height; i++)
		for (int j = 0; j < Width; j++)
		{
			fils[cont] = img(j, i);
			cont++;
		}
	return fils;
}

//Displays the image contained in a vector
void draw_fils(float * fils, int Width, int Height)
{
	IF tmp(Width, Height);
	int cont = 0;
	for (int i = 0; i < Height; i++)
		for (int j = 0; j < Width; j++)
		{
			tmp(j, i) = fils[cont];
			cont++;
		}
	tmp.display();
}


//Calculates the gradients of an image contained in a vector in Cuda
__global__ void dev_gauss(float *dev_fils, int Width, int Height, float *dev_result, int size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int N = Width * Height;
	int sm = size / 2;
	//while (tid < N){
	while (tid < (N - sm*Width) && tid >(sm*Width) && (tid %Width >sm) && (tid %Width)< (Width - sm)){
		float media=0.0;
		int beginning = tid - size*Width - sm;
		for (int l = 0; l < size; l++)
		{
			int tmp = beginning;
			for (int k = 0; k < size; k++)
			{
				media += dev_fils[tmp];
				tmp++;
			}
			beginning += Width;
		}
		media /= size;

		dev_result[tid] = media;
		tid += blockDim.x * gridDim.x;
	}

}

int main()
{
	IC img("ironman.bmp");
	IF red = img.get_channel(0);

	//-------------------------------------------------------------------------------
	//-----------------------------------------CUDA CODE-----------------------------
	//-------------------------------------------------------------------------------

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	int Width = red.width(), Height = red.height();
	int N = Width*Height;
	int size = 13;
	int MaxThreads = 1024;


	float * fils = create_fils(red, Width, Height);
	float * gxy = (float*)malloc(Width*Height*sizeof(float) - Width);

	float * dev_fils;	hipMalloc((void**)&dev_fils, Width*Height*sizeof(float));
	float * dev_result;	hipMalloc((void**)&dev_result, Width*Height*sizeof(float) - Width);

	hipMemcpy(dev_fils, fils, Width*Height*sizeof(float), hipMemcpyHostToDevice);

	//KERNEL EXECUTION
	hipEventRecord(start);
	dev_gauss << <(N + MaxThreads - 1) / MaxThreads, MaxThreads >> >(dev_fils, Width, Height, dev_result,size);
	hipEventRecord(stop);


	hipMemcpy(gxy, dev_result, Width*Height*sizeof(float) - Width, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);


	//PRINTING PARALLEL PROCESSING TIME: 76.123 milisegundos

	float milliseconds = 0;
	cout << "time spent" << endl;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << milliseconds << endl;
	cout << "time spent" << endl;

	//Freeing memory
	hipFree(dev_fils);
	hipFree(dev_result);


	cout << "Displaying computed image in cuda: " << endl;
	draw_fils(gxy, Width, Height - 1);

	return 0;

}
